#include <iostream>
#include <hip/hip_runtime.h>

// __global__ define uma função (kernel) que roda na GPU
__global__ void olaCUDA() {
    printf("Olá, CUDA, direto da GPU!\n");
}

int main() {
    // A sintaxe <<<1, 1>>> chama o kernel para ser executado na GPU
    olaCUDA<<<1, 1>>>();

    // Sincroniza a CPU com a GPU para aguardar o término do kernel
    hipDeviceSynchronize();

    return 0;
}
