#include <iostream>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>

__global__ void addVectors2D(float *a, float *b, float *c, int n) {
    int threadsPerBlock = blockDim.x * blockDim.y;
    int blockOffset = blockIdx.x * threadsPerBlock;
    int threadOffset = threadIdx.y * blockDim.x + threadIdx.x;
    int index = blockOffset + threadOffset;

    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void verify(const std::vector<float>& c, int n) {
    for (int i = 0; i < n; ++i) {
        if (c[i] != 3.0f * i) {
            std::cout << "Erro na verificação no índice " << i << std::endl;
            return;
        }
    }
    std::cout << "Sucesso! O resultado da GPU está correto." << std::endl;
}

int main() {
    const int n = 1048576; // 2^20
    const size_t size = n * sizeof(float);

    std::vector<float> h_a(n);
    std::vector<float> h_b(n);
    std::vector<float> h_c(n);

    for(int i = 0; i < n; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    int numBlocks = (n + (threadsPerBlock.x * threadsPerBlock.y) - 1) / (threadsPerBlock.x * threadsPerBlock.y);
    dim3 gridDim(numBlocks);

    addVectors2D<<<gridDim, threadsPerBlock>>>(d_a, d_b, d_c, n);
    
    hipMemcpy(h_c.data(), d_c, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    verify(h_c, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
