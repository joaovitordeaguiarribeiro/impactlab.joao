#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void addVectors(float *a, float *b, float *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void verifyResult(const std::vector<float>& c, const std::vector<float>& a, const std::vector<float>& b) {
    const float tolerance = 1e-5;
    for (size_t i = 0; i < c.size(); ++i) {
        if (abs(c[i] - (a[i] + b[i])) > tolerance) {
            std::cout << "Erro no índice " << i << std::endl;
            return;
        }
    }
    std::cout << "Sucesso! Os resultados da GPU estão corretos." << std::endl;
}

int main() {
    int n = 1024 * 1024;
    size_t size = n * sizeof(float);

    std::vector<float> h_a(n);
    std::vector<float> h_b(n);
    std::vector<float> h_c(n);

    for (int i = 0; i < n; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c.data(), d_c, size, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    verifyResult(h_c, h_a, h_b);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
